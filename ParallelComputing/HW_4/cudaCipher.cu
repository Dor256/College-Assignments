#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include "prototype.h"

__global__  void decryptKernel(struct Result* result, int maxKey, int fromKey, char* inputData, size_t inputLen, char* wordData, size_t wordLen) {
	for (i = fromKey; i < maxKey; i++) {
		// Get threadId
		threadId = omp_get_thread_num();
		// Get binary string representation fo key
		key = decimalToBinary(i);
		// Try to decrypt the cipher with the current key
		decrypted = encryptDecrypt(key, keyLen, inputData, inputLen);
		// Check if the decrypted plaintext makes sense by matching it with the known words text
		if (validate(decrypted, wordData, wordLen)) {
			// Print the threadId that managed to decrypt
			printf("Solving thread is: %d\n", threadId);
			// Allocate memory for encryption key & plaintext
			result->key = (char*) malloc(keyLen * sizeof(char));
			result->plaintext = (char*) malloc(MAX_TEXT_LENGTH * sizeof(char));

			// Save encryption key & plaintext
			strcpy(result->key, key);
			strcpy(result->plaintext, decrypted);
		}
	}
}


struct Result* cudaDecrypt(int maxKey, int fromKey, int keyLen, char* inputData, size_t inputLen, char* wordData, size_t wordLen) {

    // Holds error strings for cuda errors
    hipError_t err = hipSuccess;
    struct Result *result;
    char *key, *decrypted;

    // Allocate memory on GPU for data from the host
    err = hipMalloc((struct Result**)&result, sizeof(struct Result*));
    if (err != hipSuccess) {
        fprintf(stderr, "Memory allocation failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    result->key = NULL;
    result->plaintext = NULL;
    
    decryptKernel<<<1, 5>>>(result, maxKey, fromKey, inputData, inputLen, wordData, wordLen);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Call to decryptKernel failed: %s\n", hipGetErrorString(err));
    }

    // Call calculation kernel
    // calculationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, size / (threadsPerBlock * blocksPerGrid), bucket, blocksPerGrid, threadsPerBlock);
    // err = hipGetLastError();
    // if (err != hipSuccess) {
    //     fprintf(stderr, "Call to calculationKernel failed: %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed after call to calculationKernel:  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (result && result->key) {
        printf("%s", result->key);
    } else {
        printf("OK");
    }

    // Call merge kernel
    // mergeKernel<<<1, BUCKET_SIZE>>>(temp, bucket, blocksPerGrid, threadsPerBlock);
    // err = hipGetLastError();
    // if (err != hipSuccess) {
    //     fprintf(stderr, "Call to mergeKernel failed:  %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // err = hipDeviceSynchronize();
    // if (err != hipSuccess) {
    //     fprintf(stderr, "hipDeviceSynchronize failed after call to mergKernel -  %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // // Copy result from GPU to the host memory.
    // histogram = (int*) malloc(sizeof(int) * BUCKET_SIZE);
    // if (!histogram) {
    //     fprintf(stderr, "Memory allocation failed\n");
    //     exit(EXIT_FAILURE);	
    // }
    // err = hipMemcpy(histogram, temp, sizeof(int) * BUCKET_SIZE, hipMemcpyDeviceToHost);
    // if (err != hipSuccess) {
    //     fprintf(stderr, "Failed to copy from device to host: %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // Free memory on GPU
    // if (hipFree(d_A) != hipSuccess) {
    //     fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // // Free memory on GPU
    // if (hipFree(bucket) != hipSuccess) {
    //     fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }
    // // Free memory on GPU
    // if (hipFree(temp) != hipSuccess) {
    //     fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
    //     exit(EXIT_FAILURE);
    // }

    // return histogram;
}

