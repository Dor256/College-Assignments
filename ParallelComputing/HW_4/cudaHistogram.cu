#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include "prototype.h"

__global__  void calculationKernel(int *arr, int part, int *bucket, int numOfBlocks, int threadsPerBlock) {
	int threadId, blockId, start, end, startOfBucket, i;

	threadId = threadIdx.x;
	blockId = blockIdx.x;

	start = threadId * part;
	end = start + part;

	if (threadId < threadsPerBlock) {
		startOfBucket = (blockId * threadsPerBlock + threadId) * BUCKET_SIZE;
		for (i = start; i < end; ++i) {
			bucket[startOfBucket + arr[i]]++;
		}
	}
}

__global__  void mergeKernel(int *histogram, int *bucket, int numOfBlocks, int threadsPerBlock) {
	int threadId, i, j;

	threadId = threadIdx.x;

	for (i = 0; i < numOfBlocks; i++) {
		for (j = 0; j < threadsPerBlock; j++) {
			histogram[threadId] += bucket[(i * threadsPerBlock + j) * BUCKET_SIZE+ threadId];
		}
	}
}


int* cudaHistogramCalc(int *arr, int size) {

    // Holds error strings for cuda errors
    hipError_t err = hipSuccess;

    int threadsPerBlock, blocksPerGrid, *histogram;
    size_t length = size * sizeof(int);

    threadsPerBlock = 32;
    blocksPerGrid = 2;

    // Allocate memory on GPU for data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, length);
    if (err != hipSuccess) {
        fprintf(stderr, "Memory allocation failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy host data to GPU memory
    err = hipMemcpy(d_A, arr, length, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate GPU memory for each bucket
    int *bucket;
    err = hipMalloc((void **)&bucket, threadsPerBlock * blocksPerGrid * BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Memory allocation failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset bucket
    err = hipMemset(bucket, 0, threadsPerBlock * blocksPerGrid * BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free allocated memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate GPU memory for each bucket
    int *temp;
    err = hipMalloc((void **)&temp, BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset bucket
    err = hipMemset(temp, 0, BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free allocated memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Call calculation kernel
    calculationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, size / (threadsPerBlock * blocksPerGrid), bucket, blocksPerGrid, threadsPerBlock);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Call to calculationKernel failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed after call to calculationKernel:  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Call merge kernel
    mergeKernel<<<1, BUCKET_SIZE>>>(temp, bucket, blocksPerGrid, threadsPerBlock);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Call to mergeKernel failed:  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed after call to mergKernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy result from GPU to the host memory.
    histogram = (int*) malloc(sizeof(int) * BUCKET_SIZE);
    if (!histogram) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);	
    }
    err = hipMemcpy(histogram, temp, sizeof(int) * BUCKET_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy from device to host: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free memory on GPU
    if (hipFree(bucket) != hipSuccess) {
        fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free memory on GPU
    if (hipFree(temp) != hipSuccess) {
        fprintf(stderr, "Failed to free device memory: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return histogram;
}

